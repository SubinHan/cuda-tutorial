#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../Common/Timer.h"

__global__ void MatrixMulCuda( int*M, int*N, int*P, int LENGTH )
{
	int tid, tx, ty;
	tx = blockDim.x * blockIdx.x + threadIdx.x;
	ty = blockDim.y * blockIdx.y + threadIdx.y;
	tid = LENGTH * ty + tx;
	
	int Value = 0; int MVal = 0; int NVal = 0;
	
	for (int i = 0; i < LENGTH; i++)
	{
		MVal = M[ty * LENGTH + i];
		NVal = N[i * LENGTH + tx];
		Value += MVal * NVal;
	}
	P[tid] = Value;
}


int main()
{
	srand(time(NULL));
	
	int MatrixSize; int BufferSize;
	int* M; int* N; int* P;
	
	struct timeval htod_start, htod_end;
	struct timeval gpu_start, gpu_end;
	struct timeval dtoh_start, dtoh_end;
	
	int* dev_M; int* dev_N; int* dev_P;
	
	int grid_dim;
	int block_dim=8;
	
	for(int LENGTH=8; LENGTH<1025; LENGTH+=8){
		grid_dim = LENGTH / block_dim;
		MatrixSize = LENGTH * LENGTH;
		BufferSize = MatrixSize * sizeof(int);
		M = (int*)malloc(BufferSize); N = (int*)malloc(BufferSize); P = (int*)malloc(BufferSize);
		for( int i = 0; i < MatrixSize; i++)
		{
			M[i] = rand()%4; 
			N[i] = rand()%8; 
			P[i] = 0;
		}
		hipMalloc((void**)&dev_M, BufferSize);
		hipMalloc((void**)&dev_N, BufferSize);
		hipMalloc((void**)&dev_P, BufferSize);
		
		gettimeofday(&htod_start, NULL);
		hipMemcpy(dev_M, M, BufferSize, hipMemcpyHostToDevice);
		hipMemcpy(dev_N, N, BufferSize, hipMemcpyHostToDevice);
		
		gettimeofday(&htod_end, NULL);
		struct timeval htod_gap; getGapTime(&htod_start, &htod_end, &htod_gap); 
		float f_htod_gap = timevalToFloat(&htod_gap);
		
		dim3 Dg(grid_dim, 1, 1); dim3 Db(block_dim, 1, 1);
		
		gettimeofday(&gpu_start, NULL);
		MatrixMulCuda <<<Dg,Db>>> (dev_M, dev_N, dev_P, LENGTH);
		hipDeviceSynchronize();
		gettimeofday(&gpu_end, NULL);
		struct timeval gpu_gap; getGapTime(&gpu_start, &gpu_end, &gpu_gap); 
		float f_gpu_gap = timevalToFloat(&gpu_gap);
		
		gettimeofday(&dtoh_start, NULL);
		hipMemcpy(P, dev_P, BufferSize, hipMemcpyDeviceToHost);
		gettimeofday(&dtoh_end, NULL);
		struct timeval dtoh_gap; getGapTime(&htod_start, &dtoh_end, &dtoh_gap); 
		float f_dtoh_gap = timevalToFloat(&dtoh_gap);
		
		float total_gap = f_htod_gap + f_gpu_gap + f_dtoh_gap;
		
		printf("[Cuda] LENGTH = %d, total time = %.6f, htod time = %.6f, GPU time = %.6f, dtoh time = %.6f \n", 
		LENGTH, total_gap, f_htod_gap, f_gpu_gap, f_dtoh_gap);
		
		hipFree(dev_M); hipFree(dev_N); hipFree(dev_P);
		free(M); free(N); free(P);
	}
	return 0;
}
