
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 20
#define DIM 4

__global__ void add(int *a, int *b, int *c, int *d, int *e)
{
  int tid = blockIdx.x;
  e[tid] = a[tid] + b[tid] + c[tid] + d[tid];
}

int main()
{
  int size = N * sizeof(int);

  int a[N], b[N], c[N], d[N], e[N];

  int *dev_a, *dev_b, *dev_c, *dev_d, *dev_e;

  hipMalloc((void**)&dev_a, size);
  hipMalloc((void**)&dev_b, size);
  hipMalloc((void**)&dev_c, size);
  hipMalloc((void**)&dev_d, size);
  hipMalloc((void**)&dev_e, size);

  for(int i = 0; i < N; i++)
  {
    a[i] = i;
    b[i] = 10 * i;
    c[i] = 100 * i;
    d[i] = 1000 * i;
  }

  hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_c, c, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_d, d, size, hipMemcpyHostToDevice);

  add<<<N,1>>>(dev_a, dev_b, dev_c, dev_d, dev_e);

  hipMemcpy(e, dev_e, size, hipMemcpyDeviceToHost);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  hipFree(dev_d);
  hipFree(dev_e);

  for(int i = 0; i < N; i++)
  {
    printf("%d + %d + %d + %d = %d\n", a[i], b[i], c[i], d[i], e[i]);
  }
  
  getchar();

  return 1;
}