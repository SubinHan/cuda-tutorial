
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 20

__global__ void add(int *a, int *b, int *c)
{
  int tid = blockIdx.x;
  c[tid] = a[tid] + b[tid];
}

int main()
{
  int size = N * sizeof(int);

  int a[N], b[N], c[N];

  int *dev_a, *dev_b, *dev_c;

  hipMalloc((void**)&dev_a, size);
  hipMalloc((void**)&dev_b, size);
  hipMalloc((void**)&dev_c, size);

  for(int i = 0; i < N; i++)
  {
    a[i] = i;
    b[i] = 100 * i;
  }

  hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

  add<<<N,1>>>(dev_a, dev_b, dev_c);

  hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  for(int i = 0; i < N; i++)
  {
    printf("%d + %d = %d\n", a[i], b[i], c[i]);
  }

  return 1;
}