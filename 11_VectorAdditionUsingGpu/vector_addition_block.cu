
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 65536

__global__ void device_add(int *a, int *b, int *c) 
{
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void fill_array(int *data)
{
	for(int idx = 0; idx < N; idx++)
		data[idx] = idx;
}

void print_output(int *a, int *b, int *c)
{
	for(int idx = 0; idx < N; idx++)
		printf("\n %d + %d = %d", a[idx], b[idx], c[idx]);
}

__global__ void device_print_output(int *a, int *b, int *c)
{
	printf("\n %d + %d = %d", a[blockIdx.x], b[blockIdx.x], c[blockIdx.x]);
}

int main(void) {
	int *a, *b, *c;
	int *dev_a, *dev_b, *dev_c;
	int size = N * sizeof(int);
	
	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); fill_array(a);
	b = (int *)malloc(size); fill_array(b);
	c = (int *)malloc(size);
	
	hipMalloc((void **)&dev_a, size);
	hipMalloc((void **)&dev_b, size);
	hipMalloc((void **)&dev_c, size);
	
	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
	device_add<<<N,1>>>(dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
	
	//print_output(a, b, c);
	device_print_output<<<N, 1>>>(dev_a, dev_b, dev_c);
	
	free(a); free(b); free(c);
	hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
	
	return 0;
}
