
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
  const int size = 10;
  int src_data[size] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9};
  int dst_data[size] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

  printf("src_data = ");
  for(int i = 0; i < size; i++)
    printf("%d ", src_data[i]);
  printf("\n");

  int* dev_src = 0;
  int* dev_dst = 0;

  hipMalloc((void**)&dev_src, size * sizeof(int));
  hipMalloc((void**)&dev_dst, size * sizeof(int));

  hipMemcpy(dev_src, src_data, size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_dst, dev_src, size * sizeof(int), hipMemcpyDeviceToDevice);
  hipMemcpy(dst_data, dev_dst, size * sizeof(int), hipMemcpyDeviceToHost);

  hipFree(dev_src);
  hipFree(dev_dst);

  printf("dst_data = ");
  for(int i = 0; i < size; i++)
    printf("%d ", dst_data[i]);
  printf("\n");

  return 1;
}