
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void threadCountDataRace(int* nThreadCount)
{
	(*nThreadCount)++;
}
__global__ void atomicGlobal(int* nThreadCount)
{
	atomicAdd(nThreadCount, 1);
}

int main()
{
	const int nBlocks = 10000; const int nThreads = 1024;
	int nThreadCount = 0; int* dev_nThreadCount;
	
	hipMalloc((void**)&dev_nThreadCount, sizeof(int));
	
	hipMemset(dev_nThreadCount, 0, sizeof(int));
	
	threadCountDataRace<<<nBlocks,nThreads>>>(dev_nThreadCount);
	
	hipMemcpy(&nThreadCount, dev_nThreadCount, sizeof(int), hipMemcpyDeviceToHost);
	
	printf("스레드 실행 개수: %d 개 \n",nThreadCount);
	
	hipFree(dev_nThreadCount);
	
	return 0;
}
