
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define LENGTH 4

__global__ void MatrixMulCuda( int*M, int*N, int*P )
{
	int tid, tx, ty;
	tx = blockDim.x * blockIdx.x + threadIdx.x;
	ty = blockDim.y * blockIdx.y + threadIdx.y;
	
	int DimX = gridDim.x * blockDim.x;
	tid = DimX * ty + tx;
	
	int Value = 0; int MVal = 0; int NVal = 0;
	for (int k = 0; k < LENGTH; k++)
	{
		MVal = M[ty * DimX + k];
		NVal = N[k * DimX + tx];
		Value += MVal * NVal;
	}
	
	P[tid] = Value;
}

void printMatrix(int* mat)
{
	int row = 0; int col = 0;
	for (row = 0; row < LENGTH; row++)
	{
		for (col = 0; col < LENGTH; col++)
		{
			int Destindex = row * LENGTH + col;
			printf( "%2d ", mat[Destindex]);
		}
		printf( "\n");
	}
}

int main()
{
	srand(time(NULL));
	
	const int MatrixSize = LENGTH * LENGTH;
	const int BufferSize = MatrixSize * sizeof(int);
	
	int* M; int* N; int* P;
	
	M = (int*)malloc(BufferSize);
	N = (int*)malloc(BufferSize);
	P = (int*)malloc(BufferSize);
	
	for( int i = 0; i < MatrixSize; i++)
	{
		M[i] = rand()%4; 
		N[i] = rand()%8; 
		P[i] = 0;
	}
	
	int* dev_M; int* dev_N; int* dev_P;
	
	hipMalloc((void**)&dev_M, BufferSize);
	hipMalloc((void**)&dev_N, BufferSize);
	hipMalloc((void**)&dev_P, BufferSize);
	
	hipMemcpy(dev_M, M, BufferSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_N, N, BufferSize, hipMemcpyHostToDevice);
	
	dim3 Dg(3, 3, 1);
	dim3 Db(4, 4, 1);
	MatrixMulCuda <<<Dg,Db>>> (dev_M, dev_N, dev_P);
	
	hipMemcpy(P, dev_P, BufferSize, hipMemcpyDeviceToHost);
	
	printf("\n[matrix M]\n"); printMatrix(M);
	printf("\n[matrix N]\n"); printMatrix(N);
	printf("\n[matrix P]\n"); printMatrix(P);
	
	hipFree(dev_M); hipFree(dev_N); hipFree(dev_P);
	free(M); free(N); free(P);
	
	return 0;
}
